#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

const int BS=32;

__global__ void kernel(float *a, size_t N)
{
  int tid = threadIdx.x;
  __shared__ float s[BS];
  int blocks = (N+BS-1)/BS;
  float sum = 0.0f;
  for (int ib=0; ib<blocks; ib++)
  {
    int off = ib*BS+tid;
    s[tid] = a[off];
    for (int skip=16; skip>0; skip>>=1)
      if (tid+skip < N && tid < skip)
        s[tid] += s[tid+skip];
    sum += s[0];
  }
  a[0] = sum;
}


main()
{
  hipDevice_t cuDevice[4];
  hipCtx_t context[4];
  float *buffer[4];
  int dev_ids[4];
  int j=0;
  for (int i=0; i<5; i++)
  {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    fprintf (stderr, "Device %d  Compute ability = %d.%d\n", i, prop.major, prop.minor);
    if (prop.minor > 1)
    {
      dev_ids[j] = i;
      j++;
    }
  }
  size_t N = 20000000;
  float *host = (float*)malloc(N*sizeof(float));
  float sum = 0.0;
  for (int i=0; i<N; i++)
  {
    host[i] = drand48();
    sum += host[i];
  }
  for (int i=0; i<4; i++)
  {
    hipCtx_t ctx;
    hipDeviceGet(&(cuDevice[i]), dev_ids[i]);
    hipCtxCreate (&(context[i]), hipDeviceScheduleSpin, cuDevice[i]);
    // = hipMalloc(&(buffer[i]), (size_t)1000000*i);
    hipError_t result = hipMalloc(&(buffer[i]), (size_t)N*sizeof(float));
    hipMemcpyAsync(buffer[i], host, N*sizeof(float), hipMemcpyHostToDevice);
    if (result != hipSuccess)
    {
      fprintf (stderr, "Error allocating memory on device %d\n", i);
      abort();
    }
    else
      fprintf (stderr, "Device pointer = %lp\n", buffer[i]);
    hipCtxPopCurrent(&ctx);
  }
  #pragma omp parallel for
  for (int i=0; i<4; i++)
  {
    hipCtx_t ctx;
    fprintf (stderr, "Before kernel, i=%d\n", i);
    hipCtxPushCurrent (context[i]);
    dim3 dimBlock(BS);
    dim3 dimGrid(1);
    kernel<<<dimGrid,dimBlock>>>(buffer[i], N);
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
      fprintf (stderr, "CUDA error in kernel, i=%d\n", i);
    }
    float devsum;
    hipMemcpy(&devsum, buffer[i], 4, hipMemcpyDeviceToHost);
    fprintf (stderr, "Exact sum = %f  device sum = %f\n", sum, devsum);
    hipCtxPopCurrent(&ctx);
    fprintf (stderr, "After kernel, i=%d\n", i);
  }
}
