#include "hip/hip_runtime.h"
#include "multi_bspline.h"
#include "multi_bspline_create_cuda.h"

#ifndef NO_CUDA_MAIN
__constant__ float Acuda[48];
#endif

// typedef struct
// {
//   float *coefs;
//   uint3 stride;
//   float3 gridInv;
//   int num_splines;
// } multi_UBspline_3d_s_cuda;

#ifndef NO_CUDA_MAIN
multi_UBspline_3d_s_cuda*
create_multi_UBspline_3d_s_cuda (multi_UBspline_3d_s* spline)
{
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(Acuda), A_h, 48*sizeof(float), 0, hipMemcpyHostToDevice);

  multi_UBspline_3d_s_cuda *cuda_spline =
    (multi_UBspline_3d_s_cuda*) malloc (sizeof (multi_UBspline_3d_s_cuda*));
  
  cuda_spline->num_splines = spline->num_splines;

  int Nx = spline->x_grid.num+3;
  int Ny = spline->y_grid.num+3;
  int Nz = spline->z_grid.num+3;

  int N = spline->num_splines;
  if ((N%SPLINE_BLOCK_SIZE) != 0)
    N += 64 - (N%SPLINE_BLOCK_SIZE);
  cuda_spline->stride.x = Ny*Nz*N;
  cuda_spline->stride.y = Nz*N;
  cuda_spline->stride.z = N;

  cuda_spline->gridInv.x = spline->x_grid.delta_inv;
  cuda_spline->gridInv.y = spline->y_grid.delta_inv;
  cuda_spline->gridInv.z = spline->z_grid.delta_inv;

  size_t size = Nx*Ny*Nz*N*sizeof(float);

  hipMalloc((void**)&(cuda_spline->coefs), size);
  
  float *spline_buff = (float*)malloc(size);

  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++) 
	for (int isp=0; isp<spline->num_splines; isp++) {
	  spline_buff[ix*cuda_spline->stride.x +
		      iy*cuda_spline->stride.y +
		      iz*cuda_spline->stride.z + isp] =
	    spline->coefs[ix*spline->x_stride +
			  iy*spline->y_stride +
			  iz*spline->z_stride + isp];
	}
  hipMemcpy(cuda_spline->coefs, spline_buff, size, hipMemcpyHostToDevice);

  //free(spline_buff);

  return cuda_spline;
}
#endif


__global__ static void
eval_multi_multi_UBspline_3d_s_kernel 
(float *pos, float3 drInv, const float *coefs, float *vals[], uint3 strides)
{
  int block = blockIdx.x;
  int thr   = threadIdx.x;
  int ir    = blockIdx.y;
  int off   = block*SPLINE_BLOCK_SIZE+thr;

  __shared__ float *myval;
  __shared__ float abc[64];

  __shared__ float3 r;
  if (thr == 0) {
    r.x = pos[3*ir+0];
    r.y = pos[3*ir+1];
    r.z = pos[3*ir+2];
    myval = vals[ir];
  }
  __syncthreads();
  
  int3 index;
  float3 t;
  float s, sf;
  float4 tp[3];

  s = r.x * drInv.x;
  sf = floor(s);
  index.x = (int)sf;
  t.x = s - sf;

  s = r.y * drInv.y;
  sf = floor(s);
  index.y = (int)sf;
  t.y = s - sf;

  s = r.z * drInv.z;
  sf = floor(s);
  index.z = (int)sf;
  t.z = s - sf;
  
  tp[0] = make_float4(t.x*t.x*t.x, t.x*t.x, t.x, 1.0);
  tp[1] = make_float4(t.y*t.y*t.y, t.y*t.y, t.y, 1.0);
  tp[2] = make_float4(t.z*t.z*t.z, t.z*t.z, t.z, 1.0);

  __shared__ float a[4], b[4], c[4];
  if (thr < 4) {
    a[thr] = Acuda[4*thr+0]*tp[0].x + Acuda[4*thr+1]*tp[0].y + Acuda[4*thr+2]*tp[0].z + Acuda[4*thr+3]*tp[0].w;
    b[thr] = Acuda[4*thr+0]*tp[1].x + Acuda[4*thr+1]*tp[1].y + Acuda[4*thr+2]*tp[1].z + Acuda[4*thr+3]*tp[1].w;
    c[thr] = Acuda[4*thr+0]*tp[2].x + Acuda[4*thr+1]*tp[2].y + Acuda[4*thr+2]*tp[2].z + Acuda[4*thr+3]*tp[2].w;
  }
  __syncthreads();

  int i = (thr>>4)&3;
  int j = (thr>>2)&3;
  int k = (thr & 3);
  
  if (thr < 64)
    abc[thr] = a[i]*b[j]*c[k];
  __syncthreads();


  float val = 0.0;
  for (int i=0; i<4; i++) {
    for (int j=0; j<4; j++) {
      float *base = coefs + (index.x+i)*strides.x + (index.y+j)*strides.y + index.z*strides.z;
      for (int k=0; k<4; k++) 
  	val += abc[16*i+4*j+k] * base[off+k*strides.z];
    }
  }
  myval[off] = val;
}



__global__ static void
eval_multi_multi_UBspline_3d_s_vgh_kernel 
(float *pos, float3 drInv, const  float *coefs, 
 float *vals[], float *grads[], float *hess[], uint3 strides)
{
  int block = blockIdx.x;
  int thr   = threadIdx.x;
  int ir    = blockIdx.y;
  int off   = block*SPLINE_BLOCK_SIZE+threadIdx.x;

  __shared__ float *myval, *mygrad, *myhess;
  __shared__ float3 r;
  if (thr == 0) {
    r.x = pos[3*ir+0];
    r.y = pos[3*ir+1];
    r.z = pos[3*ir+2];
    myval  = vals[ir];
    mygrad = grads[ir];
    myhess = hess[ir];
  }
  __syncthreads();
  
  int3 index;
  float3 t;
  float s, sf;
  float4 tp[3];

  s = r.x * drInv.x;
  sf = floor(s);
  index.x = (int)sf;
  t.x = s - sf;

  s = r.y * drInv.y;
  sf = floor(s);
  index.y = (int)sf;
  t.y = s - sf;

  s = r.z * drInv.z;
  sf = floor(s);
  index.z = (int)sf;
  t.z = s - sf;
  
  tp[0] = make_float4(t.x*t.x*t.x, t.x*t.x, t.x, 1.0);
  tp[1] = make_float4(t.y*t.y*t.y, t.y*t.y, t.y, 1.0);
  tp[2] = make_float4(t.z*t.z*t.z, t.z*t.z, t.z, 1.0);

  // First 4 of a are value, second 4 are derivative, last four are
  // second derivative.
  __shared__ float a[12], b[12], c[12];
  if (thr < 12) {
    a[thr] = Acuda[4*thr+0]*tp[0].x + Acuda[4*thr+1]*tp[0].y + Acuda[4*thr+2]*tp[0].z + Acuda[4*thr+3]*tp[0].z;
    b[thr] = Acuda[4*thr+0]*tp[1].x + Acuda[4*thr+1]*tp[1].y + Acuda[4*thr+2]*tp[1].z + Acuda[4*thr+3]*tp[1].z;
    c[thr] = Acuda[4*thr+0]*tp[2].x + Acuda[4*thr+1]*tp[2].y + Acuda[4*thr+2]*tp[2].z + Acuda[4*thr+3]*tp[2].z;
  }
  __syncthreads();

  __shared__ float abc[640];
  int i = (thr>>4)&3;
  int j = (thr>>2)&3;
  int k = (thr & 3);

  abc[(16*i+4*j+k)+0]   = a[i+0]*b[j+0]*c[k+0]; // val
  abc[(16*i+4*j+k)+64]  = a[i+4]*b[j+0]*c[k+0]; // d/dx
  abc[(16*i+4*j+k)+128] = a[i+0]*b[j+4]*c[k+0]; // d/dy
  abc[(16*i+4*j+k)+192] = a[i+0]*b[j+0]*c[k+4]; // d/dz
  abc[(16*i+4*j+k)+256] = a[i+8]*b[j+0]*c[k+0]; // d2/dx2
  abc[(16*i+4*j+k)+320] = a[i+4]*b[j+4]*c[k+0]; // d2/dxdy
  abc[(16*i+4*j+k)+384] = a[i+4]*b[j+0]*c[k+4]; // d2/dxdz
  abc[(16*i+4*j+k)+448] = a[i+0]*b[j+8]*c[k+0]; // d2/dy2
  abc[(16*i+4*j+k)+512] = a[i+0]*b[j+4]*c[k+4]; // d2/dydz
  abc[(16*i+4*j+k)+576] = a[i+0]*b[j+0]*c[k+8]; // d2/dz2

  __syncthreads();

  float v = 0.0, g0=0.0,  g1=0.0, g2=0.0, 
    h00=0.0, h01=0.0, h02=0.0, h11=0.0, h12=0.0, h22=0.0;

  int n = 0;
  float *b0 = coefs + index.x*strides.x + index.y*strides.y + index.z*strides.z + off;
  for (int i=0; i<4; i++) {
    for (int j=0; j<4; j++) {
      float *base = b0 + i*strides.x + j*strides.y;
      for (int k=0; k<4; k++) {
	float c  = base[k*strides.z];
	v   += abc[n+0] * c;
	g0  += abc[n+1] * c;
	g1  += abc[n+2] * c;
	g2  += abc[n+3] * c;
	h00 += abc[n+4] * c;
	h01 += abc[n+5] * c;
	h02 += abc[n+6] * c;
	h11 += abc[n+7] * c;
	h12 += abc[n+8] * c;
	h22 += abc[n+9] * c;
	n += 10;
      }
    }
  }
  g0 *= drInv.x; 
  g1 *= drInv.y; 
  g2 *= drInv.z; 

  h00 *= drInv.x * drInv.x;  
  h01 *= drInv.x * drInv.y;  
  h02 *= drInv.x * drInv.z;  
  h11 *= drInv.y * drInv.y;  
  h12 *= drInv.y * drInv.z;  
  h22 *= drInv.z * drInv.z;  

  
  //  __shared__ float buff[6*SPLINE_BLOCK_SIZE];
  // Note, we can reuse abc, by replacing buff with abc.
  myval[off] = v;
  abc[3*thr+0] = g0; 
  abc[3*thr+1] = g1; 
  abc[3*thr+2] = g2; 
  __syncthreads();
  for (int i=0; i<3; i++) 
    mygrad[(3*block+i)*SPLINE_BLOCK_SIZE+thr] = abc[i*SPLINE_BLOCK_SIZE+thr]; 
  __syncthreads();

  // Write first half of Hessians
  abc[6*thr+0]  = h00;
  abc[6*thr+1]  = h01;
  abc[6*thr+2]  = h02;
  abc[6*thr+3]  = h11;
  abc[6*thr+4]  = h12;
  abc[6*thr+5]  = h22;
  __syncthreads();
  for (int i=0; i<6; i++) 
    myhess[(6*block+i)*SPLINE_BLOCK_SIZE+thr] = abc[i*SPLINE_BLOCK_SIZE+thr];
}


extern "C" void
eval_multi_multi_UBspline_3d_s_cuda (const multi_UBspline_3d_s_cuda *spline,
				     float *pos_d, float *vals_d[], int num)
{
  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(spline->num_splines/SPLINE_BLOCK_SIZE, num);

  eval_multi_multi_UBspline_3d_s_kernel<<<dimGrid,dimBlock>>>
    (pos_d, spline->gridInv, spline->coefs, vals_d, spline->stride);
}




void
test_multi_cuda2()
{
  int numWalkers = 1000;
  float *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  float *coefs, __device__ **vals_d, **grads_d, **hess_d;
  float *r_d, *r_h;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 128;
  Nx = Ny = Nz = 32;
  xs = Ny*Nz*N;
  ys = Nz*N;
  zs = N;

  // Setup Bspline coefficients
  int size = Nx*Ny*Nz*N*sizeof(float);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = drand48();

  Ugrid x_grid, y_grid, z_grid;
  x_grid.start = 0.0; x_grid.end = 1.0; x_grid.num = Nx;
  y_grid.start = 0.0; y_grid.end = 1.0; y_grid.num = Ny;
  z_grid.start = 0.0; z_grid.end = 1.0; z_grid.num = Nz;
  BCtype_s xBC, yBC, zBC;
  xBC.lCode = xBC.rCode = PERIODIC;
  yBC.lCode = yBC.rCode = PERIODIC;
  zBC.lCode = zBC.rCode = PERIODIC;
  

  multi_UBspline_3d_s *spline = 
    create_multi_UBspline_3d_s (x_grid, y_grid, z_grid, xBC, yBC, zBC, N);
  for (int i=0; i<N; i++) 
    set_multi_UBspline_3d_s (spline, i, coefs);

  multi_UBspline_3d_s_cuda *cudaspline = 
    create_multi_UBspline_3d_s_cuda (spline);

  // Setup device value storage
  int numVals = N*numWalkers*10;
  float *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(float));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(float));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(float*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(float*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(float*));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers + 3*i*N;
    hess[i]  = valBlock_d + 4*N*numWalkers + 6*i*N;
  }
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     3*numWalkers*sizeof(float));
  hipHostMalloc((void**)&(r_h), 3*numWalkers*sizeof(float));

  for (int ir=0; ir<numWalkers; ir++) {
    r_h[3*ir+0] = 0.5*drand48();
    r_h[3*ir+1] = 0.5*drand48();
    r_h[3*ir+2] = 0.5*drand48();
  }

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(N/SPLINE_BLOCK_SIZE,numWalkers);
  
  float vals_host[N], vals_cuda[N];

  // Check value
  for (int w=0; w<numWalkers; w++) {
    eval_multi_UBspline_3d_s (spline, r_h[3*w+0], r_h[3*w+1], r_h[3*w+2], vals_host);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_kernel<<<dimGrid,dimBlock>>> 
      (r_d, cudaspline->gridInv, cudaspline->coefs, vals_d, cudaspline->stride);
    hipMemcpy(vals_cuda, valBlock_d+(N*w), N*sizeof(float), hipMemcpyDeviceToHost);
    
    //for (int i=0; i<N; i++)
      fprintf (stderr, "%3i  %15.8e %15.8e\n", w, vals_host[0], vals_cuda[0]);
  }


  clock_t start, end;
  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_kernel<<<dimGrid,dimBlock>>> 
       (r_d, cudaspline->gridInv, cudaspline->coefs, vals_d, cudaspline->stride);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_vgh_kernel<<<dimGrid,dimBlock>>> 
       (r_d, cudaspline->gridInv, cudaspline->coefs, vals_d, grads_d, hess_d, cudaspline->stride);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGH Evals per second = %1.8e\n", 1.0/time);
  
  hipFree (spline->coefs);
  hipFree (valBlock_d);
  hipFree (vals_d);
  hipFree (grads_d);
  hipFree (hess_d);
  hipFree (r_d);
}
				    


static void *
test_multi_cuda(void *thread)
{
  hipSetDevice((int)(size_t)thread);
  fprintf (stderr, "In thread %p\n", thread);

  int numWalkers = 1000;
  float *coefs  ,  __device__ *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  float *coefs_d, __device__ **vals_d, **grads_d, **hess_d;
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
  		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  // Copy A to host
  hipMemcpy(Acuda, A_h, 48*sizeof(float), hipMemcpyHostToDevice); 

  float *r_d, *r_h;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 128;
  Nx = Ny = Nz = 32;
  xs = Ny*Nz*N;
  ys = Nz*N;
  zs = N;

  float3 drInv;
  drInv.x = 1.0/float(Nx);
  drInv.y = 1.0/float(Ny);
  drInv.z = 1.0/float(Nz);

  // Setup Bspline coefficients
  int size = Nx*Ny*Nz*N*sizeof(float);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = drand48();


  fprintf (stderr, "Filled in coefs.\n");
  fprintf (stderr, "size = %d\n", size);
  
  // Setup CUDA coefficients
  hipMalloc((void**)&coefs_d, 2*size);
  hipMemcpy(coefs_d, coefs, size, hipMemcpyHostToDevice);

  // Setup device value storage
  int numVals = N*numWalkers*10;
  float *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(float));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(float));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(float*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(float*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(float*));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers + 3*i*N;
    hess[i]  = valBlock_d + 4*N*numWalkers + 6*i*N;
  }
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     4*numWalkers*sizeof(float));
  hipHostMalloc((void**)&(r_h), 4*numWalkers*sizeof(float));

  for (int ir=0; ir<numWalkers; ir++) {
    r_h[4*ir+0] = 0.5*drand48();
    r_h[4*ir+1] = 0.5*drand48();
    r_h[4*ir+2] = 0.5*drand48();
  }

  uint3 strides;
  strides.x = xs;
  strides.y = ys;
  strides.z = zs;

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(N/SPLINE_BLOCK_SIZE,numWalkers);
  
  clock_t start, end;

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 4*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_kernel<<<dimGrid,dimBlock>>> 
       (r_d, drInv, coefs_d, vals_d, strides);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGH evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 4*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_vgh_kernel<<<dimGrid,dimBlock>>> 
       (r_d, drInv, coefs_d, vals_d, grads_d, hess_d, strides);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);
  
  // hipFree (valBlock_d);
  // hipFree (vals_d);
  // hipFree (coefs_d);
  // hipFree (r_d);

  return NULL;

}


#ifndef NO_CUDA_MAIN

main()
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  fprintf (stderr, "Detected %d CUDA devices.\n", deviceCount);

  // test_cuda();

  for (int device = 0; device < deviceCount; ++device) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    fprintf (stderr, "Device %d:\n", device);
    fprintf (stderr, "  Global memory:     %10d\n",
	     deviceProp.totalGlobalMem);
    fprintf (stderr, "  MultiProcessors:   %10d\n",
	     deviceProp.multiProcessorCount);
    fprintf (stderr, "  Registers:         %10d\n", 
	     deviceProp.regsPerBlock);
    fprintf (stderr, "  Constant memory:   %10d\n", 
	     deviceProp.totalConstMem);
    fprintf (stderr, "  Shared memory:     %10d\n", 
	     deviceProp.sharedMemPerBlock);
    fprintf (stderr, "  Clock rate:        %10d\n", 
	     deviceProp.clockRate);

  }

  //  test_multi_cuda((void*)0);
  test_multi_cuda2();
  fprintf (stderr, "After frees.\n");
}

#endif
