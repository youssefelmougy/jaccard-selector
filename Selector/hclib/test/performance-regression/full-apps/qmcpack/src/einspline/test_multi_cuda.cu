#include "hip/hip_runtime.h"
#include "multi_bspline.h"
#include "multi_bspline_create_cuda.h"
#include "multi_bspline_structs_cuda.h"
#include "multi_bspline_eval_cuda.h"


void
test_float_1d()
{
  int numWalkers = 1000;
  float *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  float *coefs,  **vals_d, **grads_d, **hess_d;
  float *r_d, *r_h;
  int xs, N;
  int Nx;

  N = 128*36;
  Nx = 100;
  xs = N;
  // Setup Bspline coefficients
  int size = Nx*N*sizeof(float);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int n=0; n<N; n++)
      coefs[ix*xs+ n] = drand48();

  Ugrid x_grid;
  x_grid.start = 0.0; x_grid.end = 1.0; x_grid.num = Nx;
  BCtype_s xBC;
  xBC.lCode = xBC.rCode = PERIODIC;

  multi_UBspline_1d_s *spline = 
    create_multi_UBspline_1d_s (x_grid, xBC, N);
  for (int i=0; i<N; i++) 
    set_multi_UBspline_1d_s (spline, i, coefs);

  multi_UBspline_1d_s_cuda *cudaspline = 
    create_multi_UBspline_1d_s_cuda (spline);

  // Setup device value storage
  int numVals = N*numWalkers*3;
  float *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(float));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(float));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(float*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(float*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(float*));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers   + i*N;
    hess[i]  = valBlock_d + 2*N*numWalkers + i*N;
  }
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     numWalkers*sizeof(float));
  hipHostMalloc((void**)&(r_h), numWalkers*sizeof(float));
  fprintf (stderr, "r_h = %p\n", r_h);

  for (int ir=0; ir<numWalkers; ir++) 
    r_h[ir] = 0.5*drand48();

  float vals_host[N], vals_cuda[N];

  // Check value
  for (int w=0; w<numWalkers; w++) {
    eval_multi_UBspline_1d_s (spline, r_h[w], vals_host);
    hipMemcpy(r_d, r_h, numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_1d_s_cuda (cudaspline, r_d, vals_d, numWalkers);
    hipMemcpy(vals_cuda, valBlock_d+(N*w), N*sizeof(float), hipMemcpyDeviceToHost);
    //for (int i=0; i<N; i++)
    if (w < 10)
      fprintf (stderr, "%3i  %15.8e %15.8e\n", w, vals_host[0], vals_cuda[0]);
  }


  clock_t start, end;
  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_1d_s_cuda (cudaspline, r_d, vals_d, numWalkers);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_1d_s_vgl_cuda (cudaspline, r_d, vals_d, grads_d, hess_d, numWalkers);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGL Evals per second = %1.8e\n", 1.0/time);
  
  hipFree (cudaspline->coefs);
  hipFree (valBlock_d);
  hipFree (vals_d);
  hipFree (grads_d);
  hipFree (hess_d);
  hipFree (r_d);
}



void
test_float()
{
  int numWalkers = 1024;
  float *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  float *coefs,  **vals_d, **grads_d, **hess_d;
  float *r_d, *r_h;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 256;
  Nx = Ny = Nz = 32;
  xs = Ny*Nz*N;
  ys = Nz*N;
  zs = N;

  // Setup Bspline coefficients
  int size = Nx*Ny*Nz*N*sizeof(float);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = drand48();

  Ugrid x_grid, y_grid, z_grid;
  x_grid.start = 0.0; x_grid.end = 1.0; x_grid.num = Nx;
  y_grid.start = 0.0; y_grid.end = 1.0; y_grid.num = Ny;
  z_grid.start = 0.0; z_grid.end = 1.0; z_grid.num = Nz;
  BCtype_s xBC, yBC, zBC;
  xBC.lCode = xBC.rCode = PERIODIC;
  yBC.lCode = yBC.rCode = PERIODIC;
  zBC.lCode = zBC.rCode = PERIODIC;
  

  multi_UBspline_3d_s *spline = 
    create_multi_UBspline_3d_s (x_grid, y_grid, z_grid, xBC, yBC, zBC, N);
  for (int i=0; i<N; i++) 
    set_multi_UBspline_3d_s (spline, i, coefs);

  multi_UBspline_3d_s_cuda *cudaspline = 
    create_multi_UBspline_3d_s_cuda (spline);

  // Setup device value storage
  int numVals = N*numWalkers*10;
  float *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(float));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(float));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(float*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(float*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(float*));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers + 3*i*N;
    hess[i]  = valBlock_d + 4*N*numWalkers + 6*i*N;
  }
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     3*numWalkers*sizeof(float));
  hipHostMalloc((void**)&(r_h), 3*numWalkers*sizeof(float));

  for (int ir=0; ir<numWalkers; ir++) {
    r_h[3*ir+0] = 0.5*drand48();
    r_h[3*ir+1] = 0.5*drand48();
    r_h[3*ir+2] = 0.5*drand48();
  }

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(N/SPLINE_BLOCK_SIZE,numWalkers);
  
  float vals_host[N], vals_cuda[N];

  // Check value
  for (int w=0; w<numWalkers; w++) {
    eval_multi_UBspline_3d_s (spline, r_h[3*w+0], r_h[3*w+1], r_h[3*w+2], vals_host);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_cuda (cudaspline, r_d, vals_d, numWalkers);
    hipMemcpy(vals_cuda, valBlock_d+(N*w), N*sizeof(float), hipMemcpyDeviceToHost);
    //for (int i=0; i<N; i++)
    if (w < 10)
      fprintf (stderr, "%3i  %15.8e %15.8e\n", w, vals_host[0], vals_cuda[0]);
  }


  clock_t start, end;
  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_cuda (cudaspline, r_d, vals_d, numWalkers);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_vgh_cuda (cudaspline, r_d, vals_d, grads_d, hess_d, numWalkers);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGH Evals per second = %1.8e\n", 1.0/time);
  
  hipFree (cudaspline->coefs);
  hipFree (valBlock_d);
  hipFree (vals_d);
  hipFree (grads_d);
  hipFree (hess_d);
  hipFree (r_d);
}



void
test_complex_float()
{
  int numWalkers = 1000;
  complex_float *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  complex_float *coefs,  **vals_d, **grads_d, **hess_d;
  float *Linv_d;
  float *r_d, *r_h;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 128;
  Nx = Ny = Nz = 32;
  xs = Ny*Nz*N;
  ys = Nz*N;
  zs = N;

  // Setup Bspline coefficients
  int size = Nx*Ny*Nz*N*sizeof(complex_float);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = std::complex<float>(drand48(), drand48());

  Ugrid x_grid, y_grid, z_grid;
  x_grid.start = 0.0; x_grid.end = 1.0; x_grid.num = Nx;
  y_grid.start = 0.0; y_grid.end = 1.0; y_grid.num = Ny;
  z_grid.start = 0.0; z_grid.end = 1.0; z_grid.num = Nz;
  BCtype_c xBC, yBC, zBC;
  xBC.lCode = xBC.rCode = PERIODIC;
  yBC.lCode = yBC.rCode = PERIODIC;
  zBC.lCode = zBC.rCode = PERIODIC;
  

  multi_UBspline_3d_c *spline = 
    create_multi_UBspline_3d_c (x_grid, y_grid, z_grid, xBC, yBC, zBC, N);
  for (int i=0; i<N; i++) 
    set_multi_UBspline_3d_c (spline, i, coefs);

  multi_UBspline_3d_c_cuda *cudaspline = 
    create_multi_UBspline_3d_c_cuda (spline);

  // Setup device value storage
  int numVals = N*numWalkers*10;
  complex_float *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(complex_float));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(complex_float));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(complex_float*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(complex_float*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(complex_float*));
  hipMalloc((void**)&(Linv_d), 9*sizeof(float));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers + 3*i*N;
    hess[i]  = valBlock_d + 4*N*numWalkers + 6*i*N;
  }
  float Linv[9] = { 1.0, 0.0, 0.0,  0.0, 1.0, 0.0,  0.0, 0.0, 1.0 };
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(Linv_d,  Linv,  9*sizeof(float), hipMemcpyHostToDevice);
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     3*numWalkers*sizeof(float));
  hipHostMalloc((void**)&(r_h), 3*numWalkers*sizeof(float));

  for (int ir=0; ir<numWalkers; ir++) {
    r_h[3*ir+0] = 0.5*drand48();
    r_h[3*ir+1] = 0.5*drand48();
    r_h[3*ir+2] = 0.5*drand48();
  }

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(N/SPLINE_BLOCK_SIZE,numWalkers);
  
  complex_float vals_host[N], vals_cuda[N];

  

  // Check value
  for (int w=0; w<numWalkers; w++) {
    eval_multi_UBspline_3d_c (spline, r_h[3*w+0], r_h[3*w+1], r_h[3*w+2], vals_host);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    //eval_multi_multi_UBspline_3d_c_cuda (cudaspline, r_d, vals_d, numWalkers);
    //eval_multi_multi_UBspline_3d_c_vgh_cuda (cudaspline, r_d, vals_d, grads_d, hess_d, numWalkers);
    eval_multi_multi_UBspline_3d_c_vgl_cuda (cudaspline, r_d, Linv_d, vals_d, grads_d, numWalkers, N);
    hipMemcpy(vals_cuda, valBlock_d+(N*w), N*sizeof(float), hipMemcpyDeviceToHost);
    //for (int i=0; i<N; i++)
    if (w < 10)
      fprintf (stderr, "%3i  %15.8e %15.8e  %15.8e %15.8e\n", w, 
	       vals_host[0].real(), vals_cuda[0].real(),
	       vals_host[0].imag(), vals_cuda[0].imag());
  }


  clock_t start, end;
  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_c_cuda (cudaspline, r_d, vals_d, numWalkers);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_c_vgh_cuda (cudaspline, r_d, vals_d, grads_d, hess_d, numWalkers);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGH Evals per second = %1.8e\n", 1.0/time);
  
  hipFree (cudaspline->coefs);
  hipFree (valBlock_d);
  hipFree (vals_d);
  hipFree (grads_d);
  hipFree (hess_d);
  hipFree (r_d);
}



void
test_double()
{
  int numWalkers = 1000;
  double *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  double *coefs,  **vals_d, **grads_d, **hess_d;
  double *r_d, *r_h;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 128;
  Nx = Ny = Nz = 32;
  xs = Ny*Nz*N;
  ys = Nz*N;
  zs = N;

  // Setup Bspline coefficients
  int size = Nx*Ny*Nz*N*sizeof(double);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = drand48();

  Ugrid x_grid, y_grid, z_grid;
  x_grid.start = 0.0; x_grid.end = 1.0; x_grid.num = Nx;
  y_grid.start = 0.0; y_grid.end = 1.0; y_grid.num = Ny;
  z_grid.start = 0.0; z_grid.end = 1.0; z_grid.num = Nz;
  BCtype_d xBC, yBC, zBC;
  xBC.lCode = xBC.rCode = PERIODIC;
  yBC.lCode = yBC.rCode = PERIODIC;
  zBC.lCode = zBC.rCode = PERIODIC;
  

  multi_UBspline_3d_d *spline = 
    create_multi_UBspline_3d_d (x_grid, y_grid, z_grid, xBC, yBC, zBC, N);
  for (int i=0; i<N; i++) 
    set_multi_UBspline_3d_d (spline, i, coefs);

  multi_UBspline_3d_d_cuda *cudaspline = 
    create_multi_UBspline_3d_d_cuda (spline);

  // Setup device value storage
  int numVals = N*numWalkers*10;
  double *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(double));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(double));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(double*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(double*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(double*));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers + 3*i*N;
    hess[i]  = valBlock_d + 4*N*numWalkers + 6*i*N;
  }
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     3*numWalkers*sizeof(double));
  hipHostMalloc((void**)&(r_h), 3*numWalkers*sizeof(double));

  for (int ir=0; ir<numWalkers; ir++) {
    r_h[3*ir+0] = 0.5*drand48();
    r_h[3*ir+1] = 0.5*drand48();
    r_h[3*ir+2] = 0.5*drand48();
  }

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(N/SPLINE_BLOCK_SIZE,numWalkers);
  
  double vals_host[N], vals_cuda[N];

  // Check value
  for (int w=0; w<numWalkers; w++) {
    eval_multi_UBspline_3d_d (spline, r_h[3*w+0], r_h[3*w+1], r_h[3*w+2], vals_host);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_d_cuda (cudaspline, r_d, vals_d, numWalkers);
    hipMemcpy(vals_cuda, valBlock_d+(N*w), N*sizeof(double), hipMemcpyDeviceToHost);
    //for (int i=0; i<N; i++)
    if (w < 10)
      fprintf (stderr, "%3i  %15.8e %15.8e\n", w, vals_host[0], vals_cuda[0]);
  }


  clock_t start, end;
  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_d_cuda (cudaspline, r_d, vals_d, numWalkers);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_d_vgh_cuda (cudaspline, r_d, vals_d, grads_d, hess_d, numWalkers);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGH Evals per second = %1.8e\n", 1.0/time);
  
  hipFree (cudaspline->coefs);
  hipFree (valBlock_d);
  hipFree (vals_d);
  hipFree (grads_d);
  hipFree (hess_d);
  hipFree (r_d);
}



void
test_complex_double()
{
  int numWalkers = 1000;
  complex_double *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  complex_double *coefs, **vals_d, **grads_d, **hess_d;
  double *r_d, *r_h;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 128;
  Nx = Ny = Nz = 32;
  xs = Ny*Nz*N;
  ys = Nz*N;
  zs = N;

  // Setup Bspline coefficients
  int size = Nx*Ny*Nz*N*sizeof(complex_double);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = std::complex<double>(drand48(), drand48());

  Ugrid x_grid, y_grid, z_grid;
  x_grid.start = 0.0; x_grid.end = 1.0; x_grid.num = Nx;
  y_grid.start = 0.0; y_grid.end = 1.0; y_grid.num = Ny;
  z_grid.start = 0.0; z_grid.end = 1.0; z_grid.num = Nz;
  BCtype_z xBC, yBC, zBC;
  xBC.lCode = xBC.rCode = PERIODIC;
  yBC.lCode = yBC.rCode = PERIODIC;
  zBC.lCode = zBC.rCode = PERIODIC;
  

  multi_UBspline_3d_z *spline = 
    create_multi_UBspline_3d_z (x_grid, y_grid, z_grid, xBC, yBC, zBC, N);
  for (int i=0; i<N; i++) 
    set_multi_UBspline_3d_z (spline, i, coefs);

  multi_UBspline_3d_z_cuda *cudaspline = 
    create_multi_UBspline_3d_z_cuda (spline);

  // Setup device value storage
  int numVals = N*numWalkers*10;
  complex_double *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(complex_double));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(complex_double));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(complex_double*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(complex_double*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(complex_double*));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers + 3*i*N;
    hess[i]  = valBlock_d + 4*N*numWalkers + 6*i*N;
  }
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     3*numWalkers*sizeof(double));
  hipHostMalloc((void**)&(r_h), 3*numWalkers*sizeof(double));

  for (int ir=0; ir<numWalkers; ir++) {
    r_h[3*ir+0] = 0.5*drand48();
    r_h[3*ir+1] = 0.5*drand48();
    r_h[3*ir+2] = 0.5*drand48();
  }

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(N/SPLINE_BLOCK_SIZE,numWalkers);
  
  complex_double vals_host[N], vals_cuda[N];

  // Check value
  for (int w=0; w<numWalkers; w++) {
    eval_multi_UBspline_3d_z (spline, r_h[3*w+0], r_h[3*w+1], r_h[3*w+2], vals_host);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_z_cuda (cudaspline, r_d, vals_d, numWalkers);
    hipMemcpy(vals_cuda, valBlock_d+(N*w), N*sizeof(double), hipMemcpyDeviceToHost);
    //for (int i=0; i<N; i++)
    if (w < 10)
      fprintf (stderr, "%3i  %15.8e %15.8e  %15.8e %15.8e\n", w, 
	       vals_host[0].real(), vals_cuda[0].real(),
	       vals_host[0].imag(), vals_cuda[0].imag());
  }


  clock_t start, end;
  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_z_cuda (cudaspline, r_d, vals_d, numWalkers);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_z_vgh_cuda (cudaspline, r_d, vals_d, grads_d, hess_d, numWalkers);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGH Evals per second = %1.8e\n", 1.0/time);
  
  hipFree (cudaspline->coefs);
  hipFree (valBlock_d);
  hipFree (vals_d);
  hipFree (grads_d);
  hipFree (hess_d);
  hipFree (r_d);
}



main() 
{
//   int deviceCount;
//   hipGetDeviceCount(&deviceCount);
//   int num_appropriate=0;
//   for (int device=0; device < deviceCount; ++device) {
//     hipDeviceProp_t deviceProp;
//     hipGetDeviceProperties(&deviceProp, device);
//     fprintf (stderr, "Device %d has architecture %d.%d\n",
// 	     device, deviceProp.major, deviceProp.minor);
//   }
//   hipSetDevice(0);	
  // fprintf(stderr, "Testing 1D single-precision real routines:\n");
  // test_float_1d();
  fprintf(stderr, "Testing 3D single-precision real routines:\n");
  test_float();
  // fprintf(stderr, "Testing 3D single-precision complex routines:\n");
  // test_complex_float();
  // fprintf(stderr, "Testing 3D double-precision real routines:\n");
  // test_double();
  // fprintf(stderr, "Testing 3D double-precision complex routines:\n");
  // test_complex_double();
}
