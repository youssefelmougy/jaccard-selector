#include <iostream>
#include <assert.h>
#include <hip/hip_runtime_api.h>

#include "hclib_cpp.h"

#define CHECK_CUDA(call) { \
    const hipError_t err = (call); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA ERROR @ %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err)); \
        exit(1); \
    } \
}

class test_functor {
    private:
        int *arr;
    public:
        test_functor(int *set_arr) : arr(set_arr) { }

        __host__ __device__ void operator()(int idx) {
            arr[idx] = idx;
        }
};

void validate(int *arr, int N) {
    for (int i = 0; i < N; i++) {
        if (arr[i] != i) {
            fprintf(stderr, "Error validating element %d. Expected %d but got "
                    "%d\n", i, i, arr[i]);
            exit(1);
        }
    }
}

int main(int argc, char **argv) {

    hclib::launch(&argc, argv, []() {
        hclib::place_t *root_pl = hclib::get_root_place();

        int num_toplevel;
        hclib::place_t **toplevel = hclib::get_children_of_place(root_pl,
                &num_toplevel);
        hclib::place_t *gpu_place = NULL;
        hclib::place_t *cpu_place = NULL;
        for (int i = 0; i < num_toplevel && (cpu_place == NULL ||
                    gpu_place == NULL); i++) {
            if (toplevel[i]->type == NVGPU_PLACE) {
                gpu_place = toplevel[i];
            } else {
                cpu_place = toplevel[i];
            }
        }
        assert(gpu_place && cpu_place);

        const int N = 1024;

        /******* Test on the CPU *******/
        int *arr = hclib::allocate_at<int>(cpu_place, N, 0);
        assert(arr);

        hclib::future_t *cpu_memset_event = hclib::async_memset(cpu_place, arr, 0,
                N, arr);

        loop_domain_t loop = {0, N, 1, 33};
        test_functor cpu_kernel(arr);
        hclib::future_t *cpu_kernel_event = hclib::forasync1D_future(
                (loop_domain_t *)&loop, cpu_kernel, FORASYNC_MODE_FLAT,
                cpu_place, cpu_memset_event);

        cpu_kernel_event->wait();

        validate(arr, N);

        /******* Test on the GPU using functors *******/
        int *d_arr = hclib::allocate_at<int>(gpu_place, N, 0);
        assert(d_arr);

        hclib::future_t *gpu_memset_event = hclib::async_memset(gpu_place, d_arr,
                0, N, d_arr);

        test_functor gpu_kernel(d_arr);
        hclib::future_t *gpu_kernel_event = hclib::forasync1D_future(
                (loop_domain_t *)&loop, gpu_kernel, FORASYNC_MODE_FLAT,
                gpu_place, gpu_memset_event);

        hclib::future_t *copy_event = hclib::async_copy(cpu_place, arr, gpu_place,
                d_arr, N, arr, gpu_kernel_event);
        copy_event->wait();

        validate(arr, N);

        printf("Passed!\n");
    });
    return 0;
}
